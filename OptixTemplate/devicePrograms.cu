#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include "LaunchParams.h"

using namespace osc;

namespace osc {
  
  /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
  extern "C" __constant__ LaunchParams optixLaunchParams;

  __device__ vec3f lightPos;
  
  static __forceinline__ __device__
  void *unpackPointer( uint32_t i0, uint32_t i1 )
  {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
  }

  static __forceinline__ __device__
  void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
  {
    const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
  }

  template<typename T>
  static __forceinline__ __device__ T* getPRD()
  {
      const uint32_t u0 = optixGetPayload_0();
      const uint32_t u1 = optixGetPayload_1();
      return reinterpret_cast<T*>(unpackPointer(u0, u1));
  }

  template<typename T>
  static __forceinline__ __device__ T* getHitNormal()
  {
      const uint32_t u2 = optixGetPayload_2();
      const uint32_t u3 = optixGetPayload_3();
      return reinterpret_cast<T*>(unpackPointer(u2, u3));
  }
  
  //------------------------------------------------------------------------------
  // closest hit and anyhit programs for radiance-type rays.
  //
  // Note eventually we will have to create one pair of those for each
  // ray type and each geometry type we want to render; but this
  // simple example doesn't use any actual geometries yet, so we only
  // create a single, dummy, set of them (we do have to have at least
  // one group of them to set up the SBT)
  //------------------------------------------------------------------------------
  
  extern "C" __global__ void __closesthit__empty() {

  }


  extern "C" __global__ void __closesthit__radiance_mesh()
  {
      const GeometrySBTData& geometrySbtData
          = *(const GeometrySBTData*)optixGetSbtDataPointer();

      vec3f normal, color;
      const TriangleMeshSBTData sbtData = geometrySbtData.triangle_data;
      // compute normal:
      const int   primID = optixGetPrimitiveIndex();
      const vec3i index = sbtData.index[primID];
      const vec3f& A = sbtData.vertex[index.x];
      const vec3f& B = sbtData.vertex[index.y];
      const vec3f& C = sbtData.vertex[index.z];
      normal = normalize(cross(C - A, B - A));
      color = sbtData.color;
      const float u = optixGetTriangleBarycentrics().x;
      const float v = optixGetTriangleBarycentrics().y;

      const vec3f pos = (1.f - u - v) * sbtData.vertex[index.x]
          + u * sbtData.vertex[index.y]
          + v * sbtData.vertex[index.z];
      vec3f lightDir = lightPos-pos;
      float tempcos = dot(normalize(lightDir), normal);
      tempcos = tempcos > 0 ? tempcos : 0;
      vec3f& prd = *(vec3f*)getPRD<vec3f>();

      vec3f lightVisibility = vec3f(1.0f);

      uint32_t u0, u1;
      packPointer(&lightVisibility, u0, u1);

      optixTrace(optixLaunchParams.traversable,
          pos,
          normalize(lightDir),
          1e-3f,    // tmin
          length(lightDir),  // tmax
          0.0f,   // rayTime
          OptixVisibilityMask(255),
          OPTIX_RAY_FLAG_NONE,//OPTIX_RAY_FLAG_NONE,
          SHADOW_RAY_TYPE,             // SBT offset
          RAY_TYPE_COUNT,               // SBT stride
          SHADOW_RAY_TYPE,             // missSBTIndex 
          u0, u1);
      prd = (0.2f + 0.8f * tempcos * lightVisibility) * color;
  }

  extern "C" __global__ void __closesthit__radiance_sphere()
  {
      const GeometrySBTData& geometrySbtData
          = *(const GeometrySBTData*)optixGetSbtDataPointer();

      vec3f normal, color;
      const SphereSBTData sbtData = geometrySbtData.sphere_data;
      normal = *(vec3f*)getHitNormal<vec3f>();
      color = sbtData.color;
      vec3f rayOrigin = optixGetWorldRayOrigin();
      vec3f rayDirection = optixGetWorldRayDirection();
      vec3f pos = sbtData.center + normal * sbtData.radius;
      vec3f lightDir = lightPos - pos;
      float tempcos = dot(normalize(lightDir), normal);
      tempcos = tempcos > 0 ? tempcos : 0;
      const float cosDN = 0.2f + .8f * tempcos;
      vec3f& prd = *(vec3f*)getPRD<vec3f>();

      vec3f lightVisibility = vec3f(1.0f);

      uint32_t u0, u1;
      packPointer(&lightVisibility, u0, u1);

      optixTrace(optixLaunchParams.traversable,
          pos,
          normalize(lightDir),
          1e-3f,    // tmin
          length(lightDir),  // tmax
          0.0f,   // rayTime
          OptixVisibilityMask(255),
          OPTIX_RAY_FLAG_NONE,//OPTIX_RAY_FLAG_NONE,
          SHADOW_RAY_TYPE,             // SBT offset
          RAY_TYPE_COUNT,               // SBT stride
          SHADOW_RAY_TYPE,             // missSBTIndex 
          u0, u1);

      prd = (0.2f + 0.8f * cosDN * lightVisibility) * color;
  }
  
  extern "C" __global__ void __anyhit__empty()
  { /*! for this simple example, this will remain empty */ }

  extern "C" __global__ void __anyhit__shadow()
  { 
      *getPRD<vec3f>() = vec3f(0.f);
      optixTerminateRay();
  }

  extern "C" __global__ void __intersection__empty() {
  }

  extern "C" __global__ void __intersection__sphere()
  {
      const GeometrySBTData& geometrySbtData
          = *(const GeometrySBTData*)optixGetSbtDataPointer();
      const SphereSBTData sbtData = geometrySbtData.sphere_data;

      const vec3f orig = optixGetWorldRayOrigin();
      const vec3f dir = optixGetWorldRayDirection();

      const vec3f center = sbtData.center;
      const float  radius = sbtData.radius;
      const vec3f O = orig - center;
      const float  l = 1 / length(dir);
      const vec3f D = dir * l;

      const float b = dot(O, D);
      const float c = dot(O, O) - radius * radius;
      const float disc = b * b - c;
      if (disc > 0.0f)
      {
          const float sdisc = sqrtf(disc);
          const float root1 = (-b - sdisc);
          const float        root11 = 0.0f;
          const vec3f       shading_normal = (O + (root1 + root11) * D) / radius;
          vec3f& normal = *(vec3f*)getHitNormal<vec3f>();
          normal = normalize(shading_normal);

          //TODO: passa a normal, burro

          optixReportIntersection(
              root1,      // t hit
              0,          // user hit kind
              optixGetPayload_0(), optixGetPayload_1(), optixGetPayload_2(), optixGetPayload_3()
          );
      }
  }
  
  //------------------------------------------------------------------------------
  // miss program that gets called for any ray that did not have a
  // valid intersection
  //
  // as with the anyhit/closest hit programs, in this example we only
  // need to have _some_ dummy function to set up a valid SBT
  // ------------------------------------------------------------------------------
  
  extern "C" __global__ void __miss__empty()
  {
  }

  extern "C" __global__ void __miss__radiance()
  {
    vec3f &prd = *(vec3f*)getPRD<vec3f>();

    const vec3f rayDir = optixGetWorldRayDirection();

    const vec3f color1 = vec3f(1.0f, 1.0f, 1.0f);
    const vec3f color2 = vec3f(0.8f, 0.0f, 0.8f);
    float t = 0.5f*(rayDir.y + 1.0f); 

    // set to constant white as background color
    prd = t*color2 + (1-t)*color1;
  }

  //------------------------------------------------------------------------------
  // ray gen program - the actual rendering happens in here
  //------------------------------------------------------------------------------
  extern "C" __global__ void __raygen__renderFrame()
  {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const auto &camera = optixLaunchParams.camera;

    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    vec3f pixelColorPRD = vec3f(0.f);

    vec3f hitNormal = vec3f(0.f);

    // the values we store the PRD pointer in:
    uint32_t u0, u1, u2, u3;
    packPointer(&pixelColorPRD, u0, u1);

    packPointer(&hitNormal, u2, u3);

    // normalized screen plane position, in [0,1]^2
    const vec2f screen(vec2f(ix+.5f,iy+.5f)
                       / vec2f(optixLaunchParams.frame.size));
    
    // generate ray direction
    vec3f rayDir = normalize(camera.direction
                             + (screen.x - 0.5f) * camera.horizontal
                             + (screen.y - 0.5f) * camera.vertical);

    lightPos = vec3f(0.0f, 3.0f, 0.0f);

    optixTrace(optixLaunchParams.traversable,
               camera.position,
               rayDir,
               0.f,    // tmin
               1e20f,  // tmax
               0.0f,   // rayTime
               OptixVisibilityMask( 255 ),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,             // SBT offset
               RAY_TYPE_COUNT,               // SBT stride
               SURFACE_RAY_TYPE,             // missSBTIndex 
               u0, u1, u2, u3 );

    const int r = int(255.99f*pixelColorPRD.x);
    const int g = int(255.99f*pixelColorPRD.y);
    const int b = int(255.99f*pixelColorPRD.z);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000
      | (r<<0) | (g<<8) | (b<<16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
  }
  
} // ::osc
